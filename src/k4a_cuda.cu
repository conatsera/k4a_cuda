#include "hip/hip_runtime.h"
﻿// k4a_cuda.cpp : Defines the entry point for the application.
//

#include "k4a_cuda.cuh"

__global__ void k4a_point_cloud_adjust(uint16_t* depth_data, float2* xy_table, float4* point_cloud)
{
	int i = (blockIdx.x * 512) + (threadIdx.x);

	float this_depth_data = __uint2float_rn(depth_data[i]);

	point_cloud[i].x = __fdividef((this_depth_data * xy_table[i].x), 1000.0f);
	point_cloud[i].y = __fdividef((this_depth_data * xy_table[i].y), 1000.0f);
	point_cloud[i].z = __fdividef(this_depth_data, 1000.0f);

}

void K4ALogger(void* ctx, k4a_log_level_t logLevel, const char* file, const int line, const char* message)
{
	printf("%d %s:%d %s", logLevel, file, line, message);
}

void ConstantInitFloat(float* data, int size, float val)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = val;
	}
}

void ConstantInitFloat2(float2* data, int size, float2 val)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = val;
	}
}

void ConstantInitFloat4(float4* data, int size, float4 val)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = val;
	}
}

K4A_CudaPointCloud::K4A_CudaPointCloud()
{
	bool success = true;

	k4a_device_configuration_t device_config = K4A_DEVICE_CONFIG_INIT_DISABLE_ALL;
	device_config.depth_mode = K4A_DEPTH_MODE_WFOV_2X2BINNED;

	if (k4a_device_open(K4A_DEVICE_DEFAULT, &device) != K4A_RESULT_SUCCEEDED)
	{
		printf("Open K4A Device failed\n");
		success = false;
	}

	if (k4a_device_get_calibration(device, device_config.depth_mode, device_config.color_resolution, &sensor_calibration) != K4A_RESULT_SUCCEEDED)
	{
		printf("Get depth camera calibration failed!\n");
		success = false;
	}
	else
	{
		dots = sensor_calibration.depth_camera_calibration.resolution_height * sensor_calibration.depth_camera_calibration.resolution_width;
		h_xy_table = new float2[dots];
	}

	if (success)
	{
		CreateXYTable();

		if (k4a_device_start_cameras(device, &device_config) != K4A_RESULT_SUCCEEDED)
		{
			printf("Start K4A cameras failed!\n");
		}

		if (k4abt_tracker_create(&sensor_calibration, &tracker) != K4A_RESULT_SUCCEEDED)
		{
			printf("Start tracker failed!\n");
		}
	}
}

K4A_CudaPointCloud::~K4A_CudaPointCloud()
{
	k4a_capture_release(capture);
	k4a_device_stop_cameras(device);
	k4a_device_close(device);
	delete h_xy_table;
	delete h_point_cloud;
}

void K4A_CudaPointCloud::CreateXYTable()
{
	h_xy_table = (float2*)malloc(kSize * sizeof(float2));
	ConstantInitFloat2(h_xy_table, kSize, make_float2(nanf(""), nanf("")));

	int width = sensor_calibration.depth_camera_calibration.resolution_width;
	int height = sensor_calibration.depth_camera_calibration.resolution_height;

	k4a_float2_t p;
	k4a_float3_t ray;
	int valid = 0;

	for (int y = 0, idx = 0; y < height; y++)
	{
		p.xy.y = (float)y;
		for (int x = 0; x < width; x++, idx++)
		{
			p.xy.x = (float)x;

			k4a_calibration_2d_to_3d(
				&sensor_calibration, &p, 1.f, K4A_CALIBRATION_TYPE_DEPTH, K4A_CALIBRATION_TYPE_DEPTH, &ray, &valid);

			if (valid)
			{
				h_xy_table[idx].x = ray.xyz.x;
				h_xy_table[idx].y = ray.xyz.y;
			}
		}
	}
}

void K4A_CudaPointCloud::GetCapture()
{
	k4a_device_get_capture(device, &capture, K4A_WAIT_INFINITE);
}

float4* K4A_CudaPointCloud::GeneratePointCloud()
{
	// Probe for a depth16 image
	k4a_image_t depth_image = k4a_capture_get_depth_image(capture);
	if (depth_image == nullptr)
	{
		printf("Failed to get depth image from capture\n");
	}

	int width = k4a_image_get_width_pixels(depth_image);
	int height = k4a_image_get_height_pixels(depth_image);

	uint16_t* depth_data = (uint16_t*)(void*)k4a_image_get_buffer(depth_image);

	float* h_depth_data = (float*)malloc(kSize * sizeof(float));

	h_point_cloud = (float4*)malloc(kSize * sizeof(float4));

	ConstantInitFloat4(h_point_cloud, kSize, make_float4(nanf(""), nanf(""), nanf(""), nanf("")));

	uint16_t* d_depth_data;
	hipMalloc(&d_depth_data, kSize * sizeof(uint16_t));
	float2* d_xy_table;
	hipMalloc(&d_xy_table, kSize * sizeof(float2));
	float4* d_point_cloud;
	hipMalloc(&d_point_cloud, kSize * sizeof(float4));

	hipMemcpy(d_depth_data, depth_data, kSize * sizeof(uint16_t), hipMemcpyHostToDevice);
	hipMemcpy(d_xy_table, h_xy_table, kSize * sizeof(float2), hipMemcpyHostToDevice);

	k4a_point_cloud_adjust << <512, 512 >> > (d_depth_data, d_xy_table, d_point_cloud);
	hipDeviceSynchronize();

	hipMemcpy(h_point_cloud, d_point_cloud, kSize * sizeof(float4), hipMemcpyDeviceToHost);

	hipFree(d_depth_data);
	hipFree(d_xy_table);
	hipFree(d_point_cloud);

	k4a_image_release(depth_image);

	return h_point_cloud;
}

int K4A_CudaPointCloud::GetSkeletonCount()
{
	return skeleton_count;
}

void K4A_CudaPointCloud::GetSkeletons()
{
	skeleton_count = 0;

	k4a_wait_result_t queue_capture_result = k4abt_tracker_enqueue_capture(tracker, capture, K4A_WAIT_INFINITE);
	//k4a_capture_release(sensor_capture);
	if (queue_capture_result == K4A_WAIT_RESULT_TIMEOUT)
	{
		// It should never hit timeout when K4A_WAIT_INFINITE is set.
		printf("Error! Add capture to tracker process queue timeout!\n");
	}
	else if (queue_capture_result == K4A_WAIT_RESULT_FAILED)
	{
		printf("Error! Add capture to tracker process queue failed!\n");
	}

	k4a_wait_result_t pop_frame_result = k4abt_tracker_pop_result(tracker, &body_frame, K4A_WAIT_INFINITE);
	if (pop_frame_result == K4A_WAIT_RESULT_SUCCEEDED)
	{
		size_t num_bodies = k4abt_frame_get_num_bodies(body_frame);
		printf("%zu bodies are detected!\n", num_bodies);

		for (int i = 0; i < num_bodies && i < MAX_TRACKED_SKELETONS; i++)
		{
			k4a_result_t get_body_result = k4abt_frame_get_body_skeleton(body_frame, i, &skeleton_group.skeletons[skeleton_count]);
			if (get_body_result == K4A_RESULT_SUCCEEDED)
			{
				skeleton_count++;
			}
		}

		for (int skel_id = skeleton_count; skel_id < MAX_TRACKED_SKELETONS; skel_id++) {
			skeleton_group.skeletons[skel_id] = k4abt_skeleton_t();
		}

		k4abt_frame_release(body_frame);
	}
	else if (pop_frame_result == K4A_WAIT_RESULT_TIMEOUT)
	{
		//  It should never hit timeout when K4A_WAIT_INFINITE is set.
		printf("Error! Pop body frame result timeout!\n");
	}
	else
	{
		printf("Pop body frame result failed!\n");
	}
}

k4abt_skeleton_t K4A_CudaPointCloud::GetSkeleton(int skel_id)
{
	return skeleton_group.skeletons[skel_id];
}