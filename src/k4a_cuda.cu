#include "hip/hip_runtime.h"
﻿// k4a_cuda.cpp : Defines the entry point for the application.
//

#include "k4a_cuda.cuh"

__constant__ float calibration_3d_to_3c[12];
__constant__ float calibration_3c_to_2c[15];

__forceinline__ __device__ void compute_correspondence(float4* depth_point,
									   uint32_t* color_data,
									   float4* color_depth_point,
									   uint32_t* color_pixel,
									   int2* dimensions,
									   int point)
{
	const float x = depth_point[point].x;
	const float y = depth_point[point].y;
	const float z = depth_point[point].z;

	float4 color_point;
	color_point.x = calibration_3d_to_3c[0] * x + calibration_3d_to_3c[1] * y + calibration_3d_to_3c[2] * z + calibration_3d_to_3c[9];
	color_point.y = calibration_3d_to_3c[3] * x + calibration_3d_to_3c[4] * y + calibration_3d_to_3c[5] * z + calibration_3d_to_3c[10];
	color_point.z = calibration_3d_to_3c[6] * x + calibration_3d_to_3c[7] * y + calibration_3d_to_3c[8] * z + calibration_3d_to_3c[11];

	color_depth_point[point].x = __fdividef(color_point.x, 1000.0f);
	color_depth_point[point].y = __fdividef(color_point.y, -1000.0f);
	color_depth_point[point].z = __fdividef(color_point.z, 1000.0f);

	float xy[2];
	xy[0] = color_point.x / color_point.z;
	xy[1] = color_point.y / color_point.z;

	float xp = xy[0] - calibration_3c_to_2c[10];
	float yp = xy[1] - calibration_3c_to_2c[11];

	float xp2 = xp * xp;
	float yp2 = yp * yp;
	float xyp = xp * yp;
	float rs = xp2 + yp2;

	if (!(rs > calibration_3c_to_2c[14] * calibration_3c_to_2c[14]))
	{
		float rss = rs * rs;
		float rsc = rss * rs;
		float a = 1.f + calibration_3c_to_2c[4] * rs + calibration_3c_to_2c[5] * rss + calibration_3c_to_2c[6] * rsc;
		float b = 1.f + calibration_3c_to_2c[7] * rs + calibration_3c_to_2c[8] * rss + calibration_3c_to_2c[9] * rsc;
		float bi;
		if (b != 0.f)
		{
			bi = 1.f / b;
		}
		else
		{
			bi = 1.f;
		}
		float d = a * bi;

		float xp_d = xp * d;
		float yp_d = yp * d;

		float rs_2xp2 = rs + 2.f * xp2;
		float rs_2yp2 = rs + 2.f * yp2;

		xp_d += rs_2xp2 * calibration_3c_to_2c[13] + 2.f * xyp * calibration_3c_to_2c[12];
		yp_d += rs_2yp2 * calibration_3c_to_2c[12] + 2.f * xyp * calibration_3c_to_2c[13];

		float xp_d_cx = xp_d + calibration_3c_to_2c[10];
		float yp_d_cy = yp_d + calibration_3c_to_2c[11];

		float2 color_pixel_xy;
		color_pixel_xy.x = xp_d_cx * calibration_3c_to_2c[2] + calibration_3c_to_2c[0];
		color_pixel_xy.y = yp_d_cy * calibration_3c_to_2c[3] + calibration_3c_to_2c[1];

		int pixel = (__float2int_rz(color_pixel_xy.y) * 2048) + __float2int_rz(color_pixel_xy.x);

		color_pixel[point] = color_data[pixel];
		color_depth_point[point].w = 1.0f;
	}
}

__global__ void transform_to_color_cloud(float4* point_cloud,
									uint32_t* color_data,
									float4* color_point_cloud,
									uint32_t* point_colors,
									int2* dimensions)
{
	int pixel_ratio = dimensions->x / blockDim.x;
	for (int j = 0; j < pixel_ratio; j++)
	{
		int pixel = (blockIdx.x * blockDim.x * j) + (threadIdx.x);
		compute_correspondence(point_cloud, color_data, color_point_cloud, point_colors, dimensions, pixel);
	}
}

__global__ void k4a_point_cloud_adjust(uint16_t* depth_data, float2* xy_table, float4* point_cloud)
{
	int i = (blockIdx.x * blockDim.x) + (threadIdx.x);

	float this_depth_data = __uint2float_rz(depth_data[i]);

	if (this_depth_data != nanf("") && this_depth_data != 0.0f)
	{
		point_cloud[i].x = __fdividef((this_depth_data * xy_table[i].x), 1000.0f);
		point_cloud[i].y = __fdividef((this_depth_data * xy_table[i].y), -1000.0f);
		point_cloud[i].z = __fdividef(this_depth_data, 1000.0f);
	}
}

__global__ void k4a_color_point_cloud_adjust(uint16_t* depth_data, float2* xy_table, float4* point_cloud)
{
	int i = (blockIdx.x * blockDim.x) + (threadIdx.x);

	float this_depth_data = __uint2float_rz(depth_data[i]);

	if (this_depth_data != nanf("") && this_depth_data != 0.0f)
	{
		point_cloud[i].x = (this_depth_data * xy_table[i].x);
		point_cloud[i].y = (this_depth_data * xy_table[i].y);
		point_cloud[i].z = this_depth_data;
	}
}

//__device__ uint32_t count = 0;
//__device__ bool is_last_thread_done;

__global__ void trim_and_conform_color(float4* color_point_cloud, float4* trimmed_cloud, uint32_t* color_points, uint32_t* trimmed_color_points, unsigned int* point_count)
{
		int pixel = (blockIdx.x * blockDim.x) + (threadIdx.x);
		if (color_point_cloud[pixel].w == 1.0f && color_point_cloud[pixel].z != 0.0f && color_point_cloud[pixel].x != nanf(""))
		{
			unsigned int point = atomicAdd(point_count, 1);

			trimmed_cloud[point] = color_point_cloud[pixel];
			trimmed_color_points[point] = color_points[pixel];
		}
}

__global__ void trim_and_conform(float4* point_cloud, float4* trimmed_cloud, unsigned int* point_count)
{
	int pixel = (blockIdx.x * blockDim.x) + (threadIdx.x);
	if (point_cloud[pixel].w == 1.0f && point_cloud[pixel].z != 0.0f && point_cloud[pixel].x != nanf(""))
	{
		unsigned int point = atomicAdd(point_count, 1);

		trimmed_cloud[point] = point_cloud[pixel];
	}
}

__global__ void k4a_skeleton_adjust(int skeleton_count, k4a_skeleton_group_t* skeletons, k4a_skeleton_group_t* skeletons_adjusted)
{
	for (int skel_id = 0; skel_id < skeleton_count; skel_id++)
	{
		int i = (blockIdx.x * 7) + (threadIdx.x);

		if (i < 27)
		{
			skeletons_adjusted->skeletons[skel_id].joints[i].orientation.wxyz.w = skeletons->skeletons[skel_id].joints[i].orientation.wxyz.w;
			skeletons_adjusted->skeletons[skel_id].joints[i].orientation.wxyz.x = skeletons->skeletons[skel_id].joints[i].orientation.wxyz.x;
			skeletons_adjusted->skeletons[skel_id].joints[i].orientation.wxyz.x = skeletons->skeletons[skel_id].joints[i].orientation.wxyz.y;
			skeletons_adjusted->skeletons[skel_id].joints[i].orientation.wxyz.z = skeletons->skeletons[skel_id].joints[i].orientation.wxyz.z;

			skeletons_adjusted->skeletons[skel_id].joints[i].position.xyz.x = __fdividef(skeletons->skeletons[skel_id].joints[i].position.xyz.x, -1000.0F);
			skeletons_adjusted->skeletons[skel_id].joints[i].position.xyz.y = __fdividef(skeletons->skeletons[skel_id].joints[i].position.xyz.y, -1000.0F);
			skeletons_adjusted->skeletons[skel_id].joints[i].position.xyz.z = __fdividef(skeletons->skeletons[skel_id].joints[i].position.xyz.z, 1000.0F);
		}
	}
}

void K4ALogger(void* ctx, k4a_log_level_t logLevel, const char* file, const int line, const char* message)
{
	printf("%d %s:%d %s", logLevel, file, line, message);
}

void ConstantInitFloat(float* data, int size, float val)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = val;
	}
}

void ConstantInitFloat2(float2* data, int size, float2 val)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = val;
	}
}

void ConstantInitFloat4(float4* data, int size, float4 val)
{
	for (int i = 0; i < size; ++i)
	{
		data[i] = val;
	}
}

K4A_CudaPointCloud::K4A_CudaPointCloud(bool color, bool body_tracking)
{
	bool success = true;
	color_enabled = color;

	k4a_device_configuration_t device_config = K4A_DEVICE_CONFIG_INIT_DISABLE_ALL;
	device_config.depth_mode = K4A_DEPTH_MODE_WFOV_2X2BINNED;
	if (color_enabled)
	{
		device_config.depth_mode = K4A_DEPTH_MODE_NFOV_UNBINNED;
		device_config.color_format = K4A_IMAGE_FORMAT_COLOR_BGRA32;
		device_config.color_resolution = K4A_COLOR_RESOLUTION_1536P;
		device_config.synchronized_images_only = true;
	}

	if (k4a_device_open(K4A_DEVICE_DEFAULT, &device) != K4A_RESULT_SUCCEEDED)
	{
		printf("Open K4A Device failed\n");
		success = false;
	}

	if (k4a_device_get_calibration(device, device_config.depth_mode, device_config.color_resolution, &sensor_calibration) != K4A_RESULT_SUCCEEDED)
	{
		printf("Get depth camera calibration failed!\n");
		success = false;
	}
	else
	{
		if (color_enabled)
			dots = sensor_calibration.color_camera_calibration.resolution_height* sensor_calibration.color_camera_calibration.resolution_width;
		else
			dots = sensor_calibration.depth_camera_calibration.resolution_height * sensor_calibration.depth_camera_calibration.resolution_width;
		depth_points = sensor_calibration.depth_camera_calibration.resolution_height * sensor_calibration.depth_camera_calibration.resolution_width;
	}

	if (success)
	{
		hipDevice_t cuda_device;
		hipGetDevice(&cuda_device);

		unsigned int cuda_flags = CUctx_flags::hipDeviceScheduleBlockingSync + CUctx_flags::hipDeviceMapHost;

		hipDevicePrimaryCtxSetFlags(cuda_device, cuda_flags);

		CreateXYTable();

		if (k4a_device_start_cameras(device, &device_config) != K4A_RESULT_SUCCEEDED)
		{
			printf("Start K4A cameras failed!\n");
		}

		if (body_tracking)
		{
			if (k4abt_tracker_create(&sensor_calibration, &tracker) != K4A_RESULT_SUCCEEDED)
			{
				printf("Start tracker failed!\n");
			}
		}
	}
}

K4A_CudaPointCloud::~K4A_CudaPointCloud()
{
	k4a_capture_release(capture);
	k4a_device_stop_cameras(device);
	k4a_device_close(device);
	delete h_xy_table;
	delete h_point_cloud;
}

void K4A_CudaPointCloud::CreateXYTable()
{
	h_xy_table = (float2*)malloc(depth_points * sizeof(float2));
	ConstantInitFloat2(h_xy_table, depth_points, make_float2(nanf(""), nanf("")));

	int width = sensor_calibration.depth_camera_calibration.resolution_width;
	int height = sensor_calibration.depth_camera_calibration.resolution_height;

	k4a_float2_t p;
	k4a_float3_t ray;
	int valid = 0;

	for (int y = 0, idx = 0; y < height; y++)
	{
		p.xy.y = (float)y;
		for (int x = 0; x < width; x++, idx++)
		{
			p.xy.x = (float)x;

			k4a_calibration_2d_to_3d(
					&sensor_calibration, &p, 1.f, K4A_CALIBRATION_TYPE_DEPTH, K4A_CALIBRATION_TYPE_DEPTH, &ray, &valid);

			if (valid)
			{
				h_xy_table[idx].x = ray.xyz.x;
				h_xy_table[idx].y = ray.xyz.y;
			}
		}
	}
}

void K4A_CudaPointCloud::GetCapture()
{
	if (capture != NULL)
		k4a_capture_release(capture);
	k4a_device_get_capture(device, &capture, K4A_WAIT_INFINITE);
}

float h_calibration_3d_to_3c[12];
float h_calibration_3c_to_2c[15];

int K4A_CudaPointCloud::GetMaxPointCount()
{
	return dots;
}

void K4A_CudaPointCloud::SetupPointCloud(float4** point_cloud, uint32_t** color_points)
{
	hipMalloc(&d_depth_data, depth_points * sizeof(uint16_t));
	hipMalloc(&d_xy_table, depth_points * sizeof(float2));
	hipMalloc(&d_point_cloud, dots * sizeof(float4));

	//hipMalloc(&d_trimmed_color_points, dots * sizeof(uint32_t));
	//hipMalloc(&d_trimmed_point_cloud, dots * sizeof(float4));
	hipMalloc(&d_point_count, sizeof(unsigned int));

	hipMemcpy(d_xy_table, h_xy_table, depth_points * sizeof(float2), hipMemcpyHostToDevice);

	h_point_count = (unsigned int*)malloc(sizeof(unsigned int));
	//h_point_cloud = (float4*)malloc(dots * sizeof(float4));
	h_point_cloud = (*point_cloud);
	empty_cloud = (float4*)malloc(dots * sizeof(float4));
	ConstantInitFloat4(empty_cloud, dots, make_float4(nanf(""), nanf(""), nanf(""), nanf("")));
	memcpy(h_point_cloud, empty_cloud, dots * sizeof(float4));

	hipHostRegister(h_point_cloud, sizeof(float4) * dots, hipHostRegisterMapped);

	hipHostGetDevicePointer<float4>(&d_trimmed_point_cloud, h_point_cloud, 0);

	if (color_enabled)
	{
		int width = sensor_calibration.color_camera_calibration.resolution_width;
		int height = sensor_calibration.color_camera_calibration.resolution_height;

		//h_color_points = (uint32_t*)malloc(dots * sizeof(uint32_t));
		h_color_points = (*color_points);
		h_dimensions = new int2(make_int2(width, height));

		hipHostRegister(h_color_points, sizeof(uint32_t) * dots, hipHostRegisterMapped);

		hipHostGetDevicePointer<uint32_t>(&d_trimmed_color_points, h_color_points, 0);

		hipMalloc(&d_color_data, dots * sizeof(uint32_t));
		hipMalloc(&d_color_points, dots * sizeof(uint32_t));
		hipMalloc(&d_color_point_cloud, dots * sizeof(float4));
		hipMalloc(&d_dimensions, sizeof(int2));

		hipMemcpy(d_dimensions, h_dimensions, sizeof(int2), hipMemcpyHostToDevice);

		for (int i = 0; i < 9; i++)
			h_calibration_3d_to_3c[i] = sensor_calibration.extrinsics[K4A_CALIBRATION_TYPE_DEPTH][K4A_CALIBRATION_TYPE_COLOR].rotation[i];
		for (int i = 0; i < 3; i++)
			h_calibration_3d_to_3c[i+9] = sensor_calibration.extrinsics[K4A_CALIBRATION_TYPE_DEPTH][K4A_CALIBRATION_TYPE_COLOR].translation[i];
		
		for (int i = 0; i < 14; i++)
			h_calibration_3c_to_2c[i] = sensor_calibration.color_camera_calibration.intrinsics.parameters.v[i];
		h_calibration_3c_to_2c[14] = sensor_calibration.color_camera_calibration.metric_radius;

		hipMemcpyToSymbol(HIP_SYMBOL(calibration_3d_to_3c), &h_calibration_3d_to_3c, sizeof(float) * 12);
		hipMemcpyToSymbol(HIP_SYMBOL(calibration_3c_to_2c), &h_calibration_3c_to_2c, sizeof(float) * 15);
	}
	else
	{
		h_point_cloud = (float4*)malloc(depth_points * sizeof(float4));
	}
}

void K4A_CudaPointCloud::GeneratePointCloud()
{
	// Probe for a depth16 image
	depth_image = k4a_capture_get_depth_image(capture);
	if (depth_image == nullptr)
	{
		printf("Failed to get depth image from capture\n");
	}

	if (color_enabled)
	{
		memset(h_color_points, '\0', dots * sizeof(uint32_t));

		// Probe for a depth16 image
		color_image = k4a_capture_get_color_image(capture);
		if (color_image == nullptr)
		{
			printf("Failed to get color image from capture\n");
		}
	}

	h_depth_data = (uint16_t*)(void*)k4a_image_get_buffer(depth_image);

	(*h_point_count) = 0;

	memcpy(h_point_cloud, empty_cloud, dots * sizeof(float4));

	//hipMemset(d_depth_data, 0, depth_points * sizeof(uint16_t));

	hipMemset(d_point_cloud, 0, dots * sizeof(float4));

	hipMemcpy(d_depth_data, h_depth_data, depth_points * sizeof(uint16_t), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	int blockSize = depth_points / 512;
	if (color_enabled)
		k4a_color_point_cloud_adjust << <blockSize, 512 >> > (d_depth_data, d_xy_table, d_point_cloud);
	else
		k4a_point_cloud_adjust << <blockSize, 512 >> > (d_depth_data, d_xy_table, d_point_cloud);

	if (color_enabled)
	{
		h_color_data = (uint32_t*)(void*)k4a_image_get_buffer(color_image);

		//hipMemset(d_color_data, 0, dots * sizeof(uint32_t));

		hipMemset(d_color_points, 0, dots * sizeof(uint32_t));

		hipMemset(d_color_point_cloud, 0, dots * sizeof(float4));

		hipMemcpy(d_color_data, h_color_data, dots * sizeof(uint32_t), hipMemcpyHostToDevice);

		int height = sensor_calibration.color_camera_calibration.resolution_height;

		transform_to_color_cloud<<<height, 768>>>(d_point_cloud, d_color_data, d_color_point_cloud, d_color_points, d_dimensions);

		k4a_image_release(color_image);
	}

	hipMemset(d_point_count, 0, sizeof(unsigned int));

	if (color_enabled)
		trim_and_conform_color<<<dots / 1024, 1024>>>(d_color_point_cloud, d_trimmed_point_cloud, d_color_points, d_trimmed_color_points, d_point_count);
	else
		trim_and_conform<<<dots / 1024, 1024>>>(d_point_cloud, d_trimmed_point_cloud, d_point_count);

	hipMemcpy(h_point_count, d_point_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	k4a_image_release(depth_image);
}

uint32_t K4A_CudaPointCloud::GetPointCount()
{
	return (*h_point_count);
}

void K4A_CudaPointCloud::ResetPointCloud()
{
	if (color_enabled)
	{
		delete h_dimensions;
		hipFree(d_color_data);
		hipFree(d_color_points);
		hipFree(d_color_point_cloud);
		hipFree(d_trimmed_color_points);
		hipFree(d_trimmed_point_cloud);
		hipFree(d_point_count);
		hipFree(d_dimensions);
	}
	hipFree(d_depth_data);
	hipFree(d_xy_table);
	hipFree(d_point_cloud);
}

int K4A_CudaPointCloud::GetSkeletonCount()
{
	return skeleton_count;
}


void K4A_CudaPointCloud::SetSkeletonGroup(k4a_skeleton_group_t* group_ref)
{
	h_skeleton_group = group_ref;

	hipHostRegister(h_skeleton_group, sizeof(k4a_skeleton_group_t), hipHostRegisterMapped);
	hipHostAlloc<k4a_skeleton_group_t>(&h_skeleton_group_unadjusted, sizeof(k4a_skeleton_group_t), hipHostMallocWriteCombined);

	hipHostGetDevicePointer<k4a_skeleton_group_t>(&d_skeleton_group, h_skeleton_group, 0);
	hipHostGetDevicePointer<k4a_skeleton_group_t>(&d_skeleton_group_unadjusted, h_skeleton_group_unadjusted, 0);
}

void K4A_CudaPointCloud::GetSkeletons()
{
	skeleton_count = 0;

	k4a_wait_result_t queue_capture_result = k4abt_tracker_enqueue_capture(tracker, capture, K4A_WAIT_INFINITE);
	//k4a_capture_release(sensor_capture);
	if (queue_capture_result == K4A_WAIT_RESULT_TIMEOUT)
	{
		// It should never hit timeout when K4A_WAIT_INFINITE is set.
		printf("Error! Add capture to tracker process queue timeout!\n");
	}
	else if (queue_capture_result == K4A_WAIT_RESULT_FAILED)
	{
		printf("Error! Add capture to tracker process queue failed!\n");
	}

	k4a_wait_result_t pop_frame_result = k4abt_tracker_pop_result(tracker, &body_frame, K4A_WAIT_INFINITE);
	if (pop_frame_result == K4A_WAIT_RESULT_SUCCEEDED)
	{
		size_t num_bodies = k4abt_frame_get_num_bodies(body_frame);
		//printf("%zu bodies are detected!\n", num_bodies);

		for (int i = 0; i < num_bodies && i < MAX_TRACKED_SKELETONS; i++)
		{
			k4a_result_t get_body_result = k4abt_frame_get_body_skeleton(body_frame, i, &h_skeleton_group_unadjusted->skeletons[skeleton_count]);
			if (get_body_result == K4A_RESULT_SUCCEEDED)
			{
				skeleton_count++;
			}
		}

		for (int skel_id = skeleton_count; skel_id < MAX_TRACKED_SKELETONS; skel_id++) {
			h_skeleton_group_unadjusted->skeletons[skel_id] = k4abt_skeleton_t();
		}

		k4a_skeleton_adjust<<<4, 7>>>(skeleton_count, d_skeleton_group_unadjusted, d_skeleton_group);
		hipDeviceSynchronize();

		k4abt_frame_release(body_frame);
	}
	else if (pop_frame_result == K4A_WAIT_RESULT_TIMEOUT)
	{
		//  It should never hit timeout when K4A_WAIT_INFINITE is set.
		printf("Error! Pop body frame result timeout!\n");
	}
	else
	{
		printf("Pop body frame result failed!\n");
	}
}